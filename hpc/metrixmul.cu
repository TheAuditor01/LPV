/*
 * Matrix Multiplication Program using CUDA
 * ========================================
 *
 * This program performs matrix multiplication (C = A * B) using CUDA.
 * It initializes two matrices A and B on the host (CPU), defines their dimensions,
 * copies them to the device (GPU), performs the multiplication on the GPU using a
 * custom CUDA kernel (`gpuMM`), and then copies the result matrix C back to the host to print.
 *
 * The kernel `gpuMM` assigns each thread to compute one element of the output matrix C.
 * The matrices are square (N x N), and N must be a multiple of BLOCK_SIZE (defined as 2).
 *
 * COMPILATION & EXECUTION (Linux/macOS with NVIDIA CUDA Toolkit):
 * ---------------------------------------------------------------
 * 1. Ensure NVIDIA CUDA Toolkit is installed.
 * 2. Save the code as metrixmul.cu (or metrixmul.cpp if nvcc is configured).
 * 3. Compile:
 *    nvcc metrixmul.cu -o metrixmul
 * 4. Execute:
 *    ./metrixmul
 *    The program will prompt for a value K, where matrix size N = K * BLOCK_SIZE.
 *    (Note: The code currently hardcodes K=1, so N=2).
 *
 * THEORETICAL CONCEPTS:
 *
 * CUDA (Compute Unified Device Architecture):
 * ------------------------------------------
 * - A parallel computing platform and programming model by NVIDIA for GPGPU.
 *
 * Key CUDA Concepts Used:
 * -----------------------
 * - `__global__` function (Kernel): `gpuMM` runs on the GPU.
 * - `blockIdx`, `blockDim`, `threadIdx`: Built-in variables for thread identification
 *   within a 2D grid of 2D thread blocks.
 *   - `blockIdx.y`, `blockIdx.x`: Y and X indices of the current thread block.
 *   - `blockDim.y`, `blockDim.x`: Dimensions of a thread block.
 *   - `threadIdx.y`, `threadIdx.x`: Y and X indices of the current thread within its block.
 * - `cudaMalloc()`: Allocates GPU memory.
 * - `cudaMemcpy()`: Copies data between host and device memory.
 *   - `cudaMemcpyHostToDevice`: Host to Device.
 *   - `cudaMemcpyDeviceToHost`: Device to Host.
 * - Kernel Launch (`gpuMM<<<grid,threadBlock>>>`): Executes the kernel.
 *   - `grid`: Defines the dimensions of the grid of thread blocks (KxK blocks).
 *   - `threadBlock`: Defines the dimensions of each thread block (BLOCK_SIZExBLOCK_SIZE threads).
 * - `dim3`: A CUDA data type for specifying dimensions (e.g., for grids and blocks).
 * - `cudaFree()`: Frees GPU memory (implicitly called at program termination for allocated memory if not explicitly called, but good practice to include).
 *
 * Matrix Indexing:
 * ----------------
 * - Matrices are stored in row-major order in a 1D array. Element (row, col) of an N x N
 *   matrix `M` is accessed as `M[row * N + col]`.
 *
 * Workflow:
 * ---------
 * 1. Get matrix dimension factor K from user (currently overridden to K=1, N=2).
 * 2. Initialize matrices hA, hB on the host.
 * 3. Allocate memory dA, dB, dC on the device.
 * 4. Copy hA to dA, hB to dB.
 * 5. Define grid and thread block dimensions.
 * 6. Launch `gpuMM` kernel on the device.
 * 7. Copy result dC from device to host matrix C.
 * 8. Print input and result matrices.
 * 9. Free host memory (device memory is also implicitly freed, but explicit `cudaFree` is better).
 *
 * Error Handling (Simplified):
 * ----------------------------
 * - Basic example without explicit CUDA error checking. Real applications need robust error handling.
 *
 * Printing Bug Note:
 * ------------------
 * - The printing loops for input matrices `hA` and `hB`, and the result matrix `C`,
 *   use `hA[row*col]`, `hB[row*col]`, and `C[row*col]` for indexing. This is incorrect
 *   for standard row-major or column-major matrix element access. It should be
 *   `hA[row*N + col]`, `hB[row*N + col]`, and `C[row*N + col]` respectively to print correctly.
 *   The GPU computation uses the correct `A[row*N+n]` and `B[n*N+col]` indexing.
 *
 * Sample Output (for K=1, N=2, with printing bug fixed, hA elements = 2, hB elements = 4):
 * -----------------------------------------------------------------------------------------
 * Enter a Value for Size/2 of matrix (user input, but K is set to 1)
 *
 *  Executing Matrix Multiplcation
 *
 *  Matrix size: 2x2
 *
 *  Input Matrix 1
 * 2 2
 * 2 2
 *
 *  Input Matrix 2
 * 4 4
 * 4 4
 *
 *
 *
 *
 *
 *
 *  Resultant matrix
 *
 * 16 16
 * 16 16
 * Finished.
 */
#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;
#define BLOCK_SIZE 2
__global__ void gpuMM(float *A, float *B, float *C, int N)
{
    // Matrix multiplication for NxN matrices C=A*B
    // Each thread computes a single element of C
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.f;
    for (int n = 0; n < N; ++n)
        sum += A[row * N + n] * B[n * N + col];
    C[row * N + col] = sum;
}
int main(int argc, char *argv[])
{
    int N;
    float K;
    // Perform matrix multiplication C = A*B
    // where A, B and C are NxN matrices
    // Restricted to matrices where N = K*BLOCK_SIZE;
    cout << "Enter a Value for Size/2 of matrix";
    cin >> K;
    K = 1;
    N = K * BLOCK_SIZE;
    cout << "\n Executing Matrix Multiplcation" << endl;
    cout << "\n Matrix size: " << N << "x" << N << endl;
    // Allocate memory on the host
    float *hA, *hB, *hC;
    hA = new float[N * N];
    hB = new float[N * N];
    hC = new float[N * N];
    // Initialize matrices on the host
    for (int j = 0; j < N; j++)
    {
        for (int i = 0; i < N; i++)
        {
            hA[j * N + i] = 2;
            hB[j * N + i] = 4;
        }
    } // Allocate memory on the device
    int size = N * N * sizeof(float); // Size of the memory in bytes
    float *dA, *dB, *dC;
    hipMalloc(&dA, size);
    hipMalloc(&dB, size);
    hipMalloc(&dC, size);
    dim3 threadBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(K, K);
    cout << "\n Input Matrix 1 \n";
    for (int row = 0; row < N; row++)
    {
        for (int col = 0; col < N; col++)
        {
            cout << hA[row * N + col] << " ";
        }
        cout << endl;
    }
    cout << "\n Input Matrix 2 \n";
    for (int row = 0; row < N; row++)
    {
        for (int col = 0; col < N; col++)
        {
            cout << hB[row * N + col] << " ";
        }
        cout << endl;
    }
    // Copy matrices from the host to device
    hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, size, hipMemcpyHostToDevice);
    // Execute the matrix multiplication kernel
    gpuMM<<<grid, threadBlock>>>(dA, dB, dC, N);
    // Now do the matrix multiplication on the CPU
    /*float sum;
    for (int row=0; row<N; row++){
    for (int col=0; col<N; col++){
    sum = 0.f;
    for (int n=0; n<N; n++){
    sum += hA[row*N+n]*hB[n*N+col];
    }
    hC[row*N+col] = sum;
    cout << sum <<" ";
    }
    cout<<endl;
    }*/
    // Allocate memory to store the GPU answer on the host
    float *C;
    C = new float[N * N];
    // Now copy the GPU result back to CPU
    hipMemcpy(C, dC, size, hipMemcpyDeviceToHost);
    // Check the result and make sure it is correct
    cout << "\n\n\n\n\n Resultant matrix\n\n";
    for (int row = 0; row < N; row++)
    {
        for (int col = 0; col < N; col++)
        {
            cout << C[row * N + col] << " ";
        }
        cout << endl;
    }
    cout << "Finished." << endl;

    // Free host memory
    delete[] hA;
    delete[] hB;
    delete[] C; // This was hC in the original thought, but C is what holds GPU result.
                // hC is currently unused if CPU block is commented.

    // Free device memory
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    return 0; // Ensure main returns 0
}

/*
 * ## Conceptual Overview of the CUDA Matrix Multiplication Program ##
 *
 * This program demonstrates how to perform matrix multiplication (C = A * B)
 * using NVIDIA's CUDA platform, leveraging the parallel processing power of a GPU.
 *
 * 1. Matrix Multiplication Basics:
 *    - If A is an (m x p) matrix and B is a (p x n) matrix, their product C will be an (m x n) matrix.
 *    - Each element C[row][col] is calculated as the dot product of the 'row'-th row of A
 *      and the 'col'-th column of B.
 *    - C[row][col] = sum(A[row][k] * B[k][col]) for k from 0 to p-1.
 *    - In this program, all matrices are N x N (square).
 *
 * 2. CUDA Parallelization Strategy:
 *    - The core idea is to assign the computation of each element of the result matrix C
 *      to a separate CUDA thread.
 *    - GPU threads are organized into a grid of thread blocks. For a 2D matrix, it's
 *      natural to use a 2D grid of 2D thread blocks.
 *    - `threadIdx.x`, `threadIdx.y`: Give the coordinates of a thread within its block.
 *    - `blockIdx.x`, `blockIdx.y`: Give the coordinates of a thread block within the grid.
 *    - `blockDim.x`, `blockDim.y`: Give the dimensions of a thread block.
 *    - The global row and column in the matrix that a thread is responsible for is calculated as:
 *        `int row = blockIdx.y * blockDim.y + threadIdx.y;`
 *        `int col = blockIdx.x * blockDim.x + threadIdx.x;`
 *    - `dim3 threadBlock(BLOCK_SIZE, BLOCK_SIZE);` defines each block to have BLOCK_SIZE x BLOCK_SIZE threads.
 *    - `dim3 grid(K, K);` defines the grid to have K x K blocks. Since N = K * BLOCK_SIZE, the total
 *      number of threads matches the N x N elements of the matrix.
 *
 * 3. Memory Management:
 *    - Host (CPU) Memory: Matrices `hA`, `hB` are initially created and populated on the CPU's RAM
 *      using `new float[N*N]`.
 *    - Device (GPU) Memory: Memory for matrices `dA`, `dB`, `dC` on the GPU's RAM is allocated
 *      using `cudaMalloc()`.
 *    - Data Transfers:
 *        - `cudaMemcpy(dA, hA, size, cudaMemcpyHostToDevice);` copies matrix A from host to device.
 *        - `cudaMemcpy(C, dC, size, cudaMemcpyDeviceToHost);` copies the result matrix C from device back to host.
 *    - Deallocation: It's crucial to free allocated memory to prevent leaks.
 *        - `delete[]` for host memory allocated with `new[]`.
 *        - `cudaFree()` for device memory allocated with `cudaMalloc()`.
 *
 * 4. The Kernel (`gpuMM` function):
 *    - `__global__ void gpuMM(...)`: This CUDA C/C++ keyword signifies that `gpuMM` is a kernel function.
 *      It's executed on the GPU and can be called from the host (CPU) code.
 *    - Each thread executing this kernel first determines the `row` and `col` of the element
 *      of matrix C it is responsible for.
 *    - It then iterates (the `for (int n = 0; n < N; ++n)` loop) to compute the sum of products
 *      `A[row*N+n] * B[n*N+col]`, which is the value for `C[row*N+col]`.
 *    - Matrices are stored linearly in memory (row-major order), so `A[row][n]` is accessed as `A[row*N+n]`.
 *
 * 5. Kernel Launch:
 *    - `gpuMM<<<grid, threadBlock>>>(dA, dB, dC, N);`
 *    - This is the syntax for launching the kernel. `grid` specifies the number of blocks in the grid,
 *      and `threadBlock` specifies the number of threads in each block.
 *
 * 6. Limitations and Considerations in this Program:
 *    - Matrix Size Constraint: The current setup assumes N is a multiple of BLOCK_SIZE.
 *      More robust code would handle arbitrary matrix sizes, possibly by adding checks within
 *      the kernel to ensure threads don't access out-of-bounds memory.
 *    - Input `K`: The user is prompted for `K`, but it's then hardcoded to `K=1`. This should be made consistent.
 *    - Error Checking: Production CUDA code should always check the return status of CUDA API calls
 *      (e.g., `cudaMalloc`, `cudaMemcpy`, kernel launches) for errors.
 *    - Shared Memory Optimization: For more complex matrix multiplications, especially larger ones,
 *      using shared memory (a fast, on-chip memory accessible by threads within the same block)
 *      can significantly improve performance by reducing global memory accesses. This involves loading
 *      tiles of the input matrices into shared memory.
 *
 * This example provides a fundamental understanding of GPU-accelerated matrix multiplication.
 * More advanced implementations would involve optimizations like tiling (using shared memory),
 * handling non-square matrices, and more sophisticated error handling.
 */