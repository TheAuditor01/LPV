/*
 * Vector Addition Program using CUDA
 * ==================================
 *
 * This program demonstrates a simple vector addition using CUDA.
 * It initializes two vectors A and B on the host (CPU), copies them to the device (GPU),
 * performs the addition on the GPU using a custom CUDA kernel, and then copies the result
 * vector C back to the host to print.
 *
 * COMPILATION & EXECUTION (Linux/macOS with NVIDIA CUDA Toolkit):
 * ---------------------------------------------------------------
 * 1. Ensure NVIDIA CUDA Toolkit is installed.
 * 2. Save the code as vectoradd.cu (or vectoradd.cpp if your nvcc is configured for it).
 * 3. Compile:
 *    nvcc vectoradd.cu -o vectoradd
 * 4. Execute:
 *    ./vectoradd
 *
 * THEORETICAL CONCEPTS:
 *
 * CUDA (Compute Unified Device Architecture):
 * ------------------------------------------
 * - A parallel computing platform and programming model created by NVIDIA.
 * - Allows software developers to use a CUDA-enabled graphics processing unit (GPU) for
 *   general purpose processing (an approach known as GPGPU).
 *
 * Key CUDA Concepts Used:
 * -----------------------
 * - `__global__` function (Kernel): A function that runs on the GPU and can be called
 *   from host code. In this program, `add` is the kernel.
 * - `blockIdx`, `blockDim`, `threadIdx`: Built-in CUDA variables that specify the unique
 *   ID of a thread within the GPU grid structure.
 *   - `blockIdx.x`: The x-dimension index of the current thread block within the grid.
 *   - `blockDim.x`: The number of threads in the x-dimension of a block.
 *   - `threadIdx.x`: The x-dimension index of the current thread within its block.
 * - `cudaMalloc()`: Allocates memory on the GPU.
 * - `cudaMemcpy()`: Copies data between host (CPU) memory and device (GPU) memory.
 *   - `cudaMemcpyHostToDevice`: Host to Device.
 *   - `cudaMemcpyDeviceToHost`: Device to Host.
 * - Kernel Launch (`add<<<blocksPerGrid, threadsPerBlock>>>`): Syntax to execute a
 *   `__global__` function on the GPU.
 *   - `blocksPerGrid`: The number of thread blocks in the grid.
 *   - `threadsPerBlock`: The number of threads in each block.
 * - `cudaFree()`: Frees memory on the GPU.
 *
 * Host Code vs. Device Code:
 * --------------------------
 * - Host code runs on the CPU (e.g., `main` function, memory allocation with `new`).
 * - Device code runs on the GPU (e.g., the `add` kernel).
 *
 * Workflow:
 * ---------
 * 1. Initialize data on the host.
 * 2. Allocate memory on the device for this data.
 * 3. Copy data from host to device.
 * 4. Execute the kernel on the device to process the data.
 * 5. Copy results from device back to host.
 * 6. Free device memory.
 * 7. Free host memory.
 *
 * Error Handling (Simplified):
 * ----------------------------
 * - This basic example does not include explicit CUDA error checking (e.g., checking return
 *   values of `cudaMalloc`, `cudaMemcpy`, `cudaGetLastError`). In real-world applications,
 *   robust error handling is crucial.
 *
 * Sample Output (for N=4, random numbers):
 * ----------------------------------------
 * Vector A: 3 6 7 5
 * Vector B: 9 2 4 1
 * Addition: 12 8 11 6
 *
 * (Note: The actual numbers will vary due to `rand() % 10`)
 */

#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;
__global__ void add(int *A, int *B, int *C, int size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size)
    {
        C[tid] = A[tid] + B[tid];
    }
}
void initialize(int *vector, int size)
{
    for (int i = 0; i < size; i++)
    {
        vector[i] = rand() % 10;
    }
}
void print(int *vector, int size)
{
    for (int i = 0; i < size; i++)
    {
        cout << vector[i] << " ";
    }
    cout << endl;
}
int main()
{
    int N = 4;
    int *A, *B, *C;
    int vectorSize = N;
    size_t vectorBytes = vectorSize * sizeof(int);
    A = new int[vectorSize];
    B = new int[vectorSize];
    C = new int[vectorSize];
    initialize(A, vectorSize);
    initialize(B, vectorSize);
    cout << "Vector A: ";
    print(A, N);
    cout << "Vector B: ";
    print(B, N);
    int *X, *Y, *Z;
    hipMalloc(&X, vectorBytes);
    hipMalloc(&Y, vectorBytes);
    hipMalloc(&Z, vectorBytes);
    hipMemcpy(X, A, vectorBytes, hipMemcpyHostToDevice);
    hipMemcpy(Y, B, vectorBytes, hipMemcpyHostToDevice);
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    add<<<blocksPerGrid, threadsPerBlock>>>(X, Y, Z, N);
    hipMemcpy(C, Z, vectorBytes, hipMemcpyDeviceToHost);
    cout << "Addition: ";
    print(C, N);
    delete[] A;
    delete[] B;
    delete[] C;
    hipFree(X);
    hipFree(Y);
    hipFree(Z);
    return 0;
}

/*
 * ## Theoretical Concepts for CUDA Vector Addition ##
 *
 * This program implements vector addition (C = A + B) using NVIDIA's CUDA framework.
 * The core idea is to leverage the parallel processing capabilities of a GPU.
 *
 * 1. Vector Addition:
 *    - Given two vectors A and B of the same size n, their sum C is a vector where each
 *      element C[i] = A[i] + B[i].
 *    - This operation is inherently parallel because each element C[i] can be computed
 *      independently of the others.
 *
 * 2. CUDA Parallelism Model:
 *    - Kernel (`__global__` function): A function written in CUDA C/C++ that runs on the GPU.
 *      In this program, `add` is the kernel.
 *    - Threads: The basic unit of execution on the GPU. Many threads run the same kernel code
 *      in parallel.
 *    - Thread Blocks: Threads are grouped into blocks. Threads within a block can cooperate by
 *      sharing data through shared memory and synchronizing their execution.
 *    - Grid: Blocks are organized into a grid. All threads in a grid execute the same kernel.
 *    - Thread Hierarchy (Built-in Variables):
 *        - `threadIdx.x`: The index of a thread within its block (1D in this case).
 *        - `blockDim.x`: The number of threads in a block (1D).
 *        - `blockIdx.x`: The index of a block within the grid (1D).
 *    - Global Thread ID: A unique ID for each thread across the entire grid can be calculated.
 *      For a 1D grid of 1D blocks (as used here):
 *      `int tid = blockIdx.x * blockDim.x + threadIdx.x;`
 *      This `tid` is then used to map each thread to a specific element of the vectors.
 *
 * 3. Memory Spaces:
 *    - Host Memory: CPU's RAM. Vectors A, B, and C are initially created here.
 *    - Device Memory: GPU's RAM. Vectors X, Y, and Z (corresponding to A, B, C) are stored here
 *      during the GPU computation.
 *    - `cudaMalloc()`: Allocates memory on the device.
 *    - `cudaMemcpy()`: Transfers data between host and device memory.
 *        - `cudaMemcpyHostToDevice`: Copies data from CPU to GPU.
 *        - `cudaMemcpyDeviceToHost`: Copies data from GPU to CPU.
 *    - `cudaFree()`: Deallocates memory on the device.
 *
 * 4. Program Workflow:
 *    a. Initialization (Host): Vectors A and B are created and filled with values on the CPU.
 *       Memory for C is also allocated on the host.
 *    b. Device Memory Allocation (Host calls CUDA API): Memory is allocated on the GPU for X, Y, Z.
 *    c. Data Transfer (Host to Device): Contents of A and B are copied to X and Y on the GPU.
 *    d. Kernel Launch (Host calls Kernel): The `add` kernel is launched on the GPU.
 *       - `add<<<blocksPerGrid, threadsPerBlock>>>(X, Y, Z, N);`
 *       - `blocksPerGrid`: Specifies the number of thread blocks in the grid.
 *       - `threadsPerBlock`: Specifies the number of threads in each block.
 *       - The host code calculates these values to ensure one thread per vector element.
 *    e. Kernel Execution (Device): Each GPU thread executes the `add` kernel. Using its unique
 *       `tid`, it computes `Z[tid] = X[tid] + Y[tid]`.
 *    f. Data Transfer (Device to Host): The resulting vector Z is copied from the GPU to vector C on the CPU.
 *    g. Cleanup (Host calls CUDA API & C++): Memory allocated on the device (`cudaFree`) and on the
 *       host (`delete[]`) is freed.
 *
 * 5. Scalability:
 *    - The performance benefit of CUDA comes from executing thousands of threads in parallel.
 *    - For vector addition, as the vector size `N` increases, the GPU can often perform the
 *      additions much faster than a CPU executing a sequential loop, provided `N` is large
 *      enough to overcome the overhead of data transfers and kernel launch.
 *
 * 6. Error Handling (Important Note):
 *    - This program omits explicit CUDA error checking (e.g., checking the return values of
 *      `cudaMalloc`, `cudaMemcpy`, and using `cudaGetLastError()` after kernel launches).
 *    - In production code, robust error handling is essential for diagnosing issues related
 *      to GPU operations or memory.
 */
